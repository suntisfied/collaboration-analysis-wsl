// minimal_cuda_test.cu
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void kernel() {
    // Empty kernel
}

int main() {
    kernel<<<1, 1>>>();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        return 1;
    } else {
        std::cout << "CUDA code ran successfully!" << std::endl;
    }
    return 0;
}
